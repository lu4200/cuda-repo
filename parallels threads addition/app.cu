#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__
void mykernel(float *A1, float *A2, float *R)
{
    int p = threadIdx.x;
    R[p] = A1[p] + A2[p];
}
 
int main()
{
    float A1[] = { 1, 2, 3, 4, 5, 6, 7, 8, 9 };
    float A2[] = { 10, 20, 30, 40, 50, 60, 70, 80, 90 };
    float R[9];
    int taille_mem = sizeof(float) * 9;
    float *a1_device;
    float *a2_device;
    float *r_device;

    hipMalloc((void**) &a1_device, taille_mem);
    hipMalloc((void**) &a2_device, taille_mem);
    hipMalloc((void**) &r_device, taille_mem);

    hipMemcpy(a1_device, A1, taille_mem, hipMemcpyHostToDevice);
    hipMemcpy(a2_device, A2, taille_mem, hipMemcpyHostToDevice);
        
    mykernel<<<1, 9>>>(a1_device, a2_device, r_device);
 	
    hipMemcpy(R, r_device, taille_mem, hipMemcpyDeviceToHost);
    //output
    for(int i = 0; i < 9; i++) {
        printf("%f\n", R[i]);
    }
}
