#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void matrixMulKernel(float *A, float *B, float *C, int rowsA, int colsA, int colsB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rowsA && col < colsB) {
        float sum = 0;
        for (int i = 0; i < colsA; ++i) {
            sum += A[row * colsA + i] * B[i * colsB + col];
        }
        C[row * colsB + col] = sum;
    }
}

void multiplyMatrices(float *A, float *B, float *C, int rowsA, int colsA, int rowsB, int colsB) {
    if (colsA != rowsB) {
        printf("Error: Matrices are not multipliable.\n");
        return;
    }

    int sizeA = rowsA * colsA * sizeof(float);
    int sizeB = rowsB * colsB * sizeof(float);
    int sizeC = rowsA * colsB * sizeof(float);

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, sizeA);
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);

    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((colsB + threadsPerBlock.x - 1) / threadsPerBlock.x, (rowsA + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixMulKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, rowsA, colsA, colsB);

    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

// int main() 
// {

// }